
#include <hip/hip_runtime.h>
__global__ void add_vectors(float *a, float *b, float *c, int n)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n)
    {
        c[idx] = a[idx] + b[idx];
    }
}
